#include "hip/hip_runtime.h"
// tot el que tingui prefix d és del dispositiu

#include "qcudarenderwindow.h"

#include "cudaviewpointinformationchannel.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cutil.h>
#include <cutil_math.h>

#include <vtkImageData.h>

#include "transferfunction.h"
#include "matrix4.h"


//////////////////////////////////////////////////////////////////////////////////// Ray cast ////////////////////////////////////////////////////////////////////////////////////



static const uint PARTITIONS = 1;  // en quants trossos es parteix la imatge en cada dimensió (per reduir col·lisions)
static const int VOLUME_MULTIPLIER = 1000000;  // multiplicador del volum a l'hora de convertir-lo en enter


// volum
static hipArray *gdVolumeArray;
texture<ushort, 3> gVolumeTexture;  // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)

// funció de transferència
static hipArray *gdTransferFunctionArray;
texture<float4, 1> gTransferFunctionTexture;    // podríem passar amb un sol float perquè només ens interessa l'opacitat: seria gaire més eficient?

// histograma
static int *gdiHistogram;
static float *gdfHistogram;

// p(O|v) * totalViewedVolume
texture<float, 1> gViewedVolumesTexture;    // textura de l'histograma de reals

// mida del volum
static uint gVolumeDataSize;
static hipExtent gVolumeDataDims;
static float3 gVolumeDims;

// per mostrar la imatge
static bool gDisplay;
static QCudaRenderWindow *gCudaRenderWindow;
static uint gRenderSize;


struct float3x4
{
    float4 f[3];
};


struct Ray
{
    float3 origin;      // origin
    float3 direction;   // direction
};


// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__ static bool intersectBox(Ray ray, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invRay = make_float3(1.0f) / ray.direction;
    float3 tbot = invRay * (boxmin - ray.origin);
    float3 ttop = invRay * (boxmax - ray.origin);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}


// transform vector by matrix (no translation)
__device__ static float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.f[0]));
    r.y = dot(v, make_float3(M.f[1]));
    r.z = dot(v, make_float3(M.f[2]));
    return r;
}


// transform vector by matrix with translation
__device__ static float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.f[0]);
    r.y = dot(v, M.f[1]);
    r.z = dot(v, M.f[2]);
    r.w = 1.0f;
    return r;
}


__device__ static uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}


__global__ void rayCastKernel(uint *image, uint imageWidth, uint imageHeight, int *histogram, hipExtent volumeDataDims, float3 volumeDims, float3x4 invViewMatrix, uint partitions, int volumeMultiplier)
{
    const int MAX_STEPS = 512;
    const float OPAQUE_ALPHA = 0.9f;
    const float RAY_STEP = 1.0f;

    float3 boxMin = make_float3(-volumeDims.x / 2.0f, -volumeDims.y / 2.0f, -volumeDims.z / 2.0f);
    float3 boxMax = make_float3(volumeDims.x / 2.0f, volumeDims.y / 2.0f, volumeDims.z / 2.0f);

    for (int pi = 0; pi < partitions; pi++)
    {
        for (int pj = 0; pj < partitions; pj++)
        {
            // píxel de la imatge on escriurem
            uint x = (__umul24(blockIdx.x, blockDim.x) + threadIdx.x) * partitions + pi;
            uint y = (__umul24(blockIdx.y, blockDim.y) + threadIdx.y) * partitions + pj;

            if (x >= imageWidth || y >= imageHeight) continue;

            // coordenades del píxel normalitzades a [-1,1]
            float u = (x / (float) (imageWidth-1)) * 2.0f - 1.0f;
            float v = (y / (float) (imageHeight-1)) * 2.0f - 1.0f;

            // calculate eye ray in world space
            Ray eyeRay;
            //eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
            //eyeRay.d = normalize(make_float3(u, v, -2.0f));
            //eyeRay.d = mul(c_invViewMatrix, eyeRay.d);
            eyeRay.origin = make_float3(mul(invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
            eyeRay.direction = normalize(make_float3(u, v, -3.0f)); // amb el 3 crec que s'assembla més a com es veu amb vtk
            eyeRay.direction = mul(invViewMatrix, eyeRay.direction);

            // find intersection with box
            float tnear, tfar;
            bool hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

            if (!hit) continue; // no hi ha intersecció

            if (tnear < 0.0f) tnear = 0.0f; // clamp to near plane

            // march along ray from front to back, accumulating color
            float4 sum = make_float4(0.0f);
            float remainingOpacity = 1.0f;
            float t = tnear;
            float maxSample = 0.0f;

            for (int i = 0; i < MAX_STEPS; i++)
            {
                float3 pos = eyeRay.origin + eyeRay.direction * t;
                //pos = pos * 0.5f + 0.5f;    // map position to [0, 1] coordinates
                pos.x = pos.x / volumeDims.x + 0.5f;
                pos.y = pos.y / volumeDims.y + 0.5f;
                pos.z = pos.z / volumeDims.z + 0.5f;
                // ara pos està a [0,1)

                // read from 3D texture
                float sample = tex3D(gVolumeTexture, pos.x, pos.y, pos.z);
                if (sample > maxSample) maxSample = sample;

                // lookup in transfer function texture
                float4 col = tex1D(gTransferFunctionTexture, sample);
                //if (col.w > maxSample) maxSample = col.w;
                //maxSample += 1.0f / MAX_STEPS;

                float volume = col.w * remainingOpacity;

                if (volume > 0.0f)
                {
                    // TODO de moment posem aquest min per controlar quan una coordenada és exactament 1, però potser hi ha una manera més bonica de fer-ho
                    int vx = min((int) (pos.x * volumeDataDims.width), volumeDataDims.width - 1);
                    int vy = min((int) (pos.y * volumeDataDims.height), volumeDataDims.height - 1);
                    int vz = min((int) (pos.z * volumeDataDims.depth), volumeDataDims.depth - 1);
                    int offset = vx + vy * volumeDataDims.width + vz * volumeDataDims.width * volumeDataDims.height;
                    //int offset = (int)(pos.x * volumeDataDims.width) + (int)(pos.y * volumeDataDims.height) * volumeDataDims.width + (int)(pos.z * volumeDataDims.depth) * volumeDataDims.width + volumeDataDims.height;

                    int iVolume = (int) (volume * volumeMultiplier);
                    atomicAdd(histogram + offset, iVolume);

                    // accumulate result
                    //sum = lerp(sum, col, col.w*density);
                    sum.x += col.x * col.w * remainingOpacity;
                    sum.y += col.y * col.w * remainingOpacity;
                    sum.z += col.z * col.w * remainingOpacity;
                    remainingOpacity *= 1.0f - col.w;
                    sum.w = 1.0f - remainingOpacity;
                    
                    if (sum.w >= OPAQUE_ALPHA) break;
                }

                t += RAY_STEP;

                if (t > tfar) break;
            }


            /*sum.x = maxSample / 1000.0f;
            sum.y = maxSample / 2000.0f;
            sum.z = maxSample / 4000.0f;
            sum.w = 1.0f;*/

            /*sum.x = maxSample;
            sum.y = maxSample;
            sum.z = maxSample;
            sum.w = 1.0f;*/

            if (image)
            {
                // write output color
                uint i = __umul24(y, imageWidth) + x;
                image[i] = rgbaFloatToInt(sum);
            }
        }   // pj
    }   // pi
}


__global__ void histogramToFloatKernel(int *iHistogram, float *fHistogram, hipExtent volumeDataDims, int volumeMultiplier)
{
    //uint i = blockIdx.x * blockDim.x + threadIdx.x;
    //if (i >= volumeDataSize) return;

    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float divisor = volumeMultiplier;
    fHistogram[i] = iHistogram[i] / divisor;
}


void cvicSetupRayCast(vtkImageData *image, const TransferFunction &transferFunction, int renderSize, int displaySize, QColor backgroundColor, bool display)
{
    ushort *data = reinterpret_cast<unsigned short*>(image->GetScalarPointer());
    gVolumeDataSize = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    gVolumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);
    double *spacing = image->GetSpacing();
    gVolumeDims = make_float3(dimensions[0] * spacing[0], dimensions[1] * spacing[1], dimensions[2] * spacing[2]);
    double *scalarRange = image->GetScalarRange();
    int rangeMax = static_cast<int>(scalarRange[1]);

    // create 3D array
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<ushort>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&gdVolumeArray, &channelDescVolumeArray, gVolumeDataDims) );

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(ushort), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams.dstArray = gdVolumeArray;
    copyParams.extent = gVolumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );

    // 3D texture parameters
    gVolumeTexture.normalized = true;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, gdVolumeArray, channelDescVolumeArray) );

    // create 1D array
    hipChannelFormatDesc channelDescTransferFunctionArray = hipCreateChannelDesc<float4>();
    CUDA_SAFE_CALL( hipMallocArray(&gdTransferFunctionArray, &channelDescTransferFunctionArray, rangeMax + 1, 1) );

    // fill 1D array
    float4 *transferFunctionTempArray = new float4[rangeMax + 1];
    for (int i = 0; i <= rangeMax; i++)
    {
        QColor color = transferFunction.getColor(i);
        double opacity = transferFunction.getOpacity(i);
        transferFunctionTempArray[i] = make_float4(color.redF(), color.greenF(), color.blueF(), opacity);
        //transferFunctionTempArray[4*i+0] = color.redF();
        //transferFunctionTempArray[4*i+1] = color.greenF();
        //transferFunctionTempArray[4*i+2] = color.blueF();
        //transferFunctionTempArray[4*i+3] = color.alphaF();
    }
    CUDA_SAFE_CALL( hipMemcpyToArray(gdTransferFunctionArray, 0, 0, reinterpret_cast<void*>(transferFunctionTempArray), (rangeMax + 1) * sizeof(float4), hipMemcpyHostToDevice) );
    delete[] transferFunctionTempArray;

    // 1D texture parameters
    //gTransferFunctionTexture.normalized = false;                    // false (predeterminat) -> [0,N) | true -> [0,1)
    //gTransferFunctionTexture.filterMode = hipFilterModePoint;      // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gTransferFunctionTexture.addressMode[0] = hipAddressModeClamp; // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)

    // bind array to 1D texture
    CUDA_SAFE_CALL( hipBindTextureToArray(gTransferFunctionTexture, gdTransferFunctionArray, channelDescTransferFunctionArray) );

    // create histogram
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&gdiHistogram), gVolumeDataSize * sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&gdfHistogram), gVolumeDataSize * sizeof(float)) );

    // histogram texture (p(O|v) * totalViewedVolume)
    CUDA_SAFE_CALL( hipBindTexture(0, gViewedVolumesTexture, reinterpret_cast<void*>(gdfHistogram), gViewedVolumesTexture.channelDesc, gVolumeDataSize * sizeof(float)) );

    // create render window
    gDisplay = display;
    if (display)
    {
        gCudaRenderWindow = new QCudaRenderWindow(backgroundColor, renderSize);
        gCudaRenderWindow->resize(displaySize, displaySize);
        gCudaRenderWindow->show();
    }
    else gCudaRenderWindow = 0;
    gRenderSize = renderSize;

    if (display)
    {
        CUDA_SAFE_CALL( cudaGLRegisterBufferObject(gCudaRenderWindow->pixelBufferObject()) );
    }
}


QVector<float> cvicRayCastAndGetHistogram(Vector3 viewpoint, Matrix4 viewMatrix)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    CUDA_SAFE_CALL( hipMemset(reinterpret_cast<void*>(gdiHistogram), 0, gVolumeDataSize * sizeof(int)) );   // buidar histograma

    // map PBO to get CUDA device pointer
    uint *pbo = 0;
    if (gDisplay)
    {
        CUDA_SAFE_CALL( cudaGLMapBufferObject(reinterpret_cast<void**>(&pbo), gCudaRenderWindow->pixelBufferObject()) );
        CUDA_SAFE_CALL( hipMemset(pbo, 0, gRenderSize * gRenderSize * sizeof(uint)) );   // això és per esborrar-lo
    }

    //CUDA_SAFE_CALL(hipMemset((void*)histogram, 0, volumeSize.width * volumeSize.height * volumeSize.depth * sizeof(ushort)));

    // call CUDA kernel, writing results to FBO
    dim3 blockSize(16, 16);
    dim3 gridSize(gRenderSize / blockSize.x / PARTITIONS, gRenderSize / blockSize.y / PARTITIONS);
    float3x4 invViewMatrix;
    /*invViewMatrix.f[0] = make_float4(modelViewMatrix[0][0], modelViewMatrix[0][1], modelViewMatrix[0][2], modelViewMatrix[0][3]);
    invViewMatrix.f[1] = make_float4(modelViewMatrix[1][0], modelViewMatrix[1][1], modelViewMatrix[1][2], modelViewMatrix[1][3]);
    invViewMatrix.f[2] = make_float4(modelViewMatrix[2][0], modelViewMatrix[2][1], modelViewMatrix[2][2], modelViewMatrix[2][3]);*/
    invViewMatrix.f[0] = make_float4(viewMatrix[0][0], viewMatrix[0][1], viewMatrix[0][2], viewpoint.x);
    invViewMatrix.f[1] = make_float4(viewMatrix[1][0], viewMatrix[1][1], viewMatrix[1][2], viewpoint.y);
    invViewMatrix.f[2] = make_float4(viewMatrix[2][0], viewMatrix[2][1], viewMatrix[2][2], viewpoint.z);
    /*std::cout << "modelViewMatrix = " << std::endl;
    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < 4; j++) std::cout << modelViewMatrix[i][j] << " ";
        std::cout << std::endl;
    }
    std::cout << "invViewMatrix =" << std::endl
              << invViewMatrix.f[0].x << " " << invViewMatrix.f[0].y << " " << invViewMatrix.f[0].z << " " << invViewMatrix.f[0].w << std::endl
              << invViewMatrix.f[1].x << " " << invViewMatrix.f[1].y << " " << invViewMatrix.f[1].z << " " << invViewMatrix.f[1].w << std::endl
              << invViewMatrix.f[2].x << " " << invViewMatrix.f[2].y << " " << invViewMatrix.f[2].z << " " << invViewMatrix.f[2].w << std::endl;*/

    rayCastKernel<<<gridSize, blockSize>>>(pbo, gRenderSize, gRenderSize, gdiHistogram, gVolumeDataDims, gVolumeDims, invViewMatrix, PARTITIONS, VOLUME_MULTIPLIER);
    //CUT_CHECK_ERROR( "kernel failed" );
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) std::cout << "ray cast kernel failed: " << hipGetErrorString(err) << std::endl;
    err = hipDeviceSynchronize();
    if (hipSuccess != err) std::cout << "sync after ray cast kernel failed: " << hipGetErrorString(err) << std::endl;


    if (gDisplay)
    {
        CUDA_SAFE_CALL( cudaGLUnmapBufferObject(gCudaRenderWindow->pixelBufferObject()) );
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop);

    if (gDisplay) gCudaRenderWindow->updateGL();



    hipEventRecord(start, 0);
    /*
    dim3 blockSize2(512);
    uint zo = gVolumeDataSize % blockSize2.x == 0 ? 0 : 1;
    dim3 gridSize2(gVolumeDataSize / blockSize2.x + zo);
    */
    dim3 blockSize2(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize2.x - 1) / blockSize2.x;
    uint blocksY = (gVolumeDataDims.height + blockSize2.y - 1) / blockSize2.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize2.z - 1) / blockSize2.z;
    dim3 gridSize2(blocksX * blocksY, blocksZ);

    histogramToFloatKernel<<<gridSize2, blockSize2>>>(gdiHistogram, gdfHistogram, gVolumeDataDims, VOLUME_MULTIPLIER);
    //CUT_CHECK_ERROR( "kernel failed" );
    /*hipError_t*/ err = hipGetLastError();
    if (hipSuccess != err) std::cout << "int->float kernel failed: " << hipGetErrorString(err) << std::endl;
    err = hipDeviceSynchronize();
    if (hipSuccess != err) std::cout << "sync after int->float kernel failed: " << hipGetErrorString(err) << std::endl;

    QVector<float> histogram(gVolumeDataSize);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(histogram.data()), reinterpret_cast<void*>(gdfHistogram), gVolumeDataSize * sizeof(float), hipMemcpyDeviceToHost) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime2 = 0.0f;
    hipEventElapsedTime(&elapsedTime2, start, stop);

    std::cout << "ray cast: " << elapsedTime1 << " ms + " << elapsedTime2 << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);    

    return histogram;
}

void cvicCleanupRayCast()
{
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(gdVolumeArray) );
    CUDA_SAFE_CALL( hipUnbindTexture(gTransferFunctionTexture) );
    CUDA_SAFE_CALL( hipFreeArray(gdTransferFunctionArray) );
    CUDA_SAFE_CALL( hipUnbindTexture(gViewedVolumesTexture) );
    CUDA_SAFE_CALL( hipFree(gdiHistogram) );
    CUDA_SAFE_CALL( hipFree(gdfHistogram) );
    if (gDisplay)
    {
        CUDA_SAFE_CALL( cudaGLUnregisterBufferObject(gCudaRenderWindow->pixelBufferObject()) );
        delete gCudaRenderWindow;
    }
}



//////////////////////////////////////////////////////////////////////////////////// p(Z) ////////////////////////////////////////////////////////////////////////////////////



static float *gdVoxelProbabilities;
texture<float, 1> gVoxelProbabilitiesTexture;


__global__ void voxelProbabilitiesKernel(float pv, float totalViewedVolume, hipExtent volumeDataDims, float *voxelProbabilities)
{
    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float pzv = tex1Dfetch(gViewedVolumesTexture, i) / totalViewedVolume;

    voxelProbabilities[i] += pv * pzv;
}


void cvicSetupVoxelProbabilities()
{
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&gdVoxelProbabilities), gVolumeDataSize * sizeof(float)) );
    CUDA_SAFE_CALL( hipMemset(reinterpret_cast<void*>(gdVoxelProbabilities), 0, gVolumeDataSize * sizeof(float)) );
    CUDA_SAFE_CALL( hipBindTexture(0, gVoxelProbabilitiesTexture, reinterpret_cast<void*>(gdVoxelProbabilities), gVoxelProbabilitiesTexture.channelDesc, gVolumeDataSize * sizeof(float)) );
}


void cvicAccumulateVoxelProbabilities( float viewProbability, float totalViewedVolume )
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Kernel

    dim3 blockSize(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize.x - 1) / blockSize.x;
    uint blocksY = (gVolumeDataDims.height + blockSize.y - 1) / blockSize.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize.z - 1) / blockSize.z;
    dim3 gridSize(blocksX * blocksY, blocksZ);

    voxelProbabilitiesKernel<<<gridSize, blockSize>>>(viewProbability, totalViewedVolume, gVolumeDataDims, gdVoxelProbabilities);
    CUT_CHECK_ERROR( "voxel probabilities kernel failed" );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "p(Z): " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


QVector<float> cvicGetVoxelProbabilities()
{
    QVector<float> voxelProbabilities( gVolumeDataSize );
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(voxelProbabilities.data()), reinterpret_cast<void*>(gdVoxelProbabilities), gVolumeDataSize * sizeof(float), hipMemcpyDeviceToHost) );
    return voxelProbabilities;
}


void cvicCleanupVoxelProbabilities()
{
    CUDA_SAFE_CALL( hipUnbindTexture(gVoxelProbabilitiesTexture) );
    CUDA_SAFE_CALL( hipFree(gdVoxelProbabilities) );
}



//////////////////////////////////////////////////////////////////////////////////// VoMI ////////////////////////////////////////////////////////////////////////////////////



static float *gdHVz = 0;
static float *gdVomi = 0;
static float3 *gdColorVomi = 0;


__global__ void HVzKernel(float pv, float totalViewedVolume, hipExtent volumeDataDims, float *HVz)
{
    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float pz = tex1Dfetch(gVoxelProbabilitiesTexture, i);
    float pzv = tex1Dfetch(gViewedVolumesTexture, i) / totalViewedVolume;
    float pvz = pv * pzv / pz;

    if (pvz > 0.0f) HVz[i] -= pvz * log2f(pvz);
}


__global__ void vomiKernel(float pv, float totalViewedVolume, hipExtent volumeDataDims, float *vomi)
{
    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float pz = tex1Dfetch(gVoxelProbabilitiesTexture, i);
    float pzv = tex1Dfetch(gViewedVolumesTexture, i) / totalViewedVolume;
    float pvz = pv * pzv / pz;

    if (pvz > 0.0f) vomi[i] += pvz * log2f(pvz / pv);
}


__global__ void colorVomiKernel(float pv, float3 color, float totalViewedVolume, hipExtent volumeDataDims, float3 *colorVomi)
{
    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float pz = tex1Dfetch(gVoxelProbabilitiesTexture, i);
    float pzv = tex1Dfetch(gViewedVolumesTexture, i) / totalViewedVolume;
    float pvz = pv * pzv / pz;

    if (pvz > 0.0f) colorVomi[i] += pvz * log2f(pvz / pv) * color;
}


void cvicSetupVomi(bool HVz, bool vomi, bool colorVomi)
{
    if (HVz)
    {
        CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gdHVz), gVolumeDataSize * sizeof(float)));
        CUDA_SAFE_CALL(hipMemset(reinterpret_cast<void*>(gdHVz), 0, gVolumeDataSize * sizeof(float)));
    }
    if (vomi)
    {
        CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gdVomi), gVolumeDataSize * sizeof(float)));
        CUDA_SAFE_CALL(hipMemset(reinterpret_cast<void*>(gdVomi), 0, gVolumeDataSize * sizeof(float)));
    }

    if (colorVomi)
    {
        CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gdColorVomi), gVolumeDataSize * sizeof(float3)));
        CUDA_SAFE_CALL(hipMemset(reinterpret_cast<void*>(gdColorVomi), 0, gVolumeDataSize * sizeof(float3)));
    }
}


void cvicAccumulateHVz(float viewProbability, float totalViewedVolume)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Kernel

    dim3 blockSize(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize.x - 1) / blockSize.x;
    uint blocksY = (gVolumeDataDims.height + blockSize.y - 1) / blockSize.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize.z - 1) / blockSize.z;
    dim3 gridSize(blocksX * blocksY, blocksZ);

    HVzKernel<<<gridSize, blockSize>>>(viewProbability, totalViewedVolume, gVolumeDataDims, gdHVz);
    CUT_CHECK_ERROR("H(V|z) kernel failed");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "H(V|z): " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void cvicAccumulateVomi(float viewProbability, float totalViewedVolume)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Kernel

    dim3 blockSize(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize.x - 1) / blockSize.x;
    uint blocksY = (gVolumeDataDims.height + blockSize.y - 1) / blockSize.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize.z - 1) / blockSize.z;
    dim3 gridSize(blocksX * blocksY, blocksZ);

    vomiKernel<<<gridSize, blockSize>>>(viewProbability, totalViewedVolume, gVolumeDataDims, gdVomi);
    CUT_CHECK_ERROR("vomi kernel failed");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "VoMI: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void cvicAccumulateColorVomi(float viewProbability, const Vector3Float &viewColor, float totalViewedVolume)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Kernel

    dim3 blockSize(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize.x - 1) / blockSize.x;
    uint blocksY = (gVolumeDataDims.height + blockSize.y - 1) / blockSize.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize.z - 1) / blockSize.z;
    dim3 gridSize(blocksX * blocksY, blocksZ);

    float3 color = make_float3(1.0f - viewColor.x, 1.0f - viewColor.y, 1.0f - viewColor.z);

    colorVomiKernel<<<gridSize, blockSize>>>(viewProbability, color, totalViewedVolume, gVolumeDataDims, gdColorVomi);
    CUT_CHECK_ERROR("color vomi kernel failed");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "CVoMI: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


QVector<float> cvicGetHVz()
{
    QVector<float> HVz(gVolumeDataSize);
    CUDA_SAFE_CALL(hipMemcpy(reinterpret_cast<void*>(HVz.data()), reinterpret_cast<void*>(gdHVz), gVolumeDataSize * sizeof(float), hipMemcpyDeviceToHost));
    return HVz;
}


QVector<float> cvicGetVomi()
{
    QVector<float> vomi(gVolumeDataSize);
    CUDA_SAFE_CALL(hipMemcpy(reinterpret_cast<void*>(vomi.data()), reinterpret_cast<void*>(gdVomi), gVolumeDataSize * sizeof(float), hipMemcpyDeviceToHost));
    return vomi;
}


QVector<Vector3Float> cvicGetColorVomi()
{
    QVector<Vector3Float> colorVomi(gVolumeDataSize);
    CUDA_SAFE_CALL(hipMemcpy(reinterpret_cast<void*>(colorVomi.data()), reinterpret_cast<void*>(gdColorVomi), gVolumeDataSize * sizeof(float3), hipMemcpyDeviceToHost));
    return colorVomi;
}


void cvicCleanupVomi()
{
    if (gdHVz)
    {
        CUDA_SAFE_CALL(hipFree(gdHVz));
    }

    if (gdVomi)
    {
        CUDA_SAFE_CALL(hipFree(gdVomi));
    }

    if (gdColorVomi)
    {
        CUDA_SAFE_CALL(hipFree(gdColorVomi));
    }
}



//////////////////////////////////////////////////////////////////////////////////// VoMI2 ////////////////////////////////////////////////////////////////////////////////////



//static float *gdHVz = 0;  // declarat a dalt
texture<float, 1> gHVzTexture;
static float *gdVomi2 = 0;


__global__ void vomi2Kernel(float HV, hipExtent volumeDataDims, float *vomi2)
{
    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float HVz = tex1Dfetch(gHVzTexture, i);

    vomi2[i] = HV - HVz;
}


QVector<float> cvicComputeVomi2(float HV, const QVector<float> &HVz)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gdHVz), gVolumeDataSize * sizeof(float)));
    CUDA_SAFE_CALL(hipMemcpy(reinterpret_cast<void*>(gdHVz), reinterpret_cast<void*>(const_cast<float*>(HVz.data())), gVolumeDataSize * sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipBindTexture(0, gHVzTexture, reinterpret_cast<void*>(gdHVz), gHVzTexture.channelDesc, gVolumeDataSize * sizeof(float)));

    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gdVomi2), gVolumeDataSize * sizeof(float)));
    CUDA_SAFE_CALL(hipMemset(reinterpret_cast<void*>(gdVomi2), 0, gVolumeDataSize * sizeof(float)));

    // Kernel

    dim3 blockSize(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize.x - 1) / blockSize.x;
    uint blocksY = (gVolumeDataDims.height + blockSize.y - 1) / blockSize.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize.z - 1) / blockSize.z;
    dim3 gridSize(blocksX * blocksY, blocksZ);

    vomi2Kernel<<<gridSize, blockSize>>>(HV, gVolumeDataDims, gdVomi2);
    CUT_CHECK_ERROR("vomi2 kernel failed");

    QVector<float> vomi2(gVolumeDataSize);
    CUDA_SAFE_CALL(hipMemcpy(reinterpret_cast<void*>(vomi2.data()), reinterpret_cast<void*>(gdVomi2), gVolumeDataSize * sizeof(float), hipMemcpyDeviceToHost));

    // Clean up

    CUDA_SAFE_CALL(hipUnbindTexture(gHVzTexture));
    CUDA_SAFE_CALL(hipFree(gdHVz));
    CUDA_SAFE_CALL(hipFree(gdVomi2));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "VoMI2: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return vomi2;
}



//////////////////////////////////////////////////////////////////////////////////// VoMI3 ////////////////////////////////////////////////////////////////////////////////////



static float *gdVomi3 = 0;


__global__ void vomi3Kernel(float pv, float totalViewedVolume, float vmi2, hipExtent volumeDataDims, float *vomi3)
{
    uint blocksX = (volumeDataDims.width + blockDim.x - 1) / blockDim.x;
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= volumeDataDims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= volumeDataDims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= volumeDataDims.depth) return;

    uint i = x + y * volumeDataDims.width + z * volumeDataDims.width * volumeDataDims.height;

    float pz = tex1Dfetch(gVoxelProbabilitiesTexture, i);
    float pzv = tex1Dfetch(gViewedVolumesTexture, i) / totalViewedVolume;
    float pvz = pv * pzv / pz;

    if (pvz > 0.0f) vomi3[i] += pvz * vmi2;
}


void cvicSetupVomi3()
{
    CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gdVomi3), gVolumeDataSize * sizeof(float)));
    CUDA_SAFE_CALL(hipMemset(reinterpret_cast<void*>(gdVomi3), 0, gVolumeDataSize * sizeof(float)));
}


void cvicAccumulateVomi3(float viewProbability, float totalViewedVolume, float vmi2)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Kernel

    dim3 blockSize(8, 8, 8);
    uint blocksX = (gVolumeDataDims.width + blockSize.x - 1) / blockSize.x;
    uint blocksY = (gVolumeDataDims.height + blockSize.y - 1) / blockSize.y;
    uint blocksZ = (gVolumeDataDims.depth + blockSize.z - 1) / blockSize.z;
    dim3 gridSize(blocksX * blocksY, blocksZ);

    vomi3Kernel<<<gridSize, blockSize>>>(viewProbability, totalViewedVolume, vmi2, gVolumeDataDims, gdVomi3);
    CUT_CHECK_ERROR("vomi3 kernel failed");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "VoMI3: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


QVector<float> cvicGetVomi3()
{
    QVector<float> vomi3(gVolumeDataSize);
    CUDA_SAFE_CALL(hipMemcpy(reinterpret_cast<void*>(vomi3.data()), reinterpret_cast<void*>(gdVomi3), gVolumeDataSize * sizeof(float), hipMemcpyDeviceToHost));
    return vomi3;
}


void cvicCleanupVomi3()
{
    CUDA_SAFE_CALL(hipFree(gdVomi3));
}
