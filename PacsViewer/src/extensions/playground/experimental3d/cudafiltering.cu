#include "hip/hip_runtime.h"
// tot el que tingui prefix d és del dispositiu

#include "cudafiltering.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_math.h>
#include <hip/hip_math_constants.h>

#include <vtkImageData.h>


//Round a / b to nearest higher integer value
__device__ __host__ int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}


texture<float, 3> gVolumeTexture;   // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)
texture<float, 3> gVolume2Texture;  // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)
texture<float, 3> gVolume3Texture;  // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)


__global__ void convolutionXKernel(float *result, float *kernel, int radius, hipExtent dims, bool texture2)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    if (!texture2)
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx + k, fy, fz) * kernel[radius - k];
    }
    else
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolume2Texture, fx + k, fy, fz) * kernel[radius - k];
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void convolutionYKernel(float *result, float *kernel, int radius, hipExtent dims, bool texture2)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    if (!texture2)
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx, fy + k, fz) * kernel[radius - k];
    }
    else
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolume2Texture, fx, fy + k, fz) * kernel[radius - k];
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void convolutionZKernel(float *result, float *kernel, int radius, hipExtent dims, bool texture2)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float sum = 0.0f;

    if (!texture2)
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolumeTexture, fx, fy, fz + k) * kernel[radius - k];
    }
    else
    {
        for (int k = -radius; k <= radius; k++) sum += tex3D(gVolume2Texture, fx, fy, fz + k) * kernel[radius - k];
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    result[i] = sum;
}


__global__ void substractionKernel(float *result, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] -= value;
}


QVector<float> cfGaussianDifference(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipEvent_t t0, t1;
    float t01 = 0.0f;
    hipEventCreate(&t0);
    hipEventCreate(&t1);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    float kernelSum = 0.0f;
    float sigma = radius / 3.0f;
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        float f = static_cast<float>(i - radius) / sigma;
        kernel[i] = expf(-f * f / 2.0f);
        kernelSum += kernel.at(i);
    }
    for (int i = 0; i < KERNEL_WIDTH; i++) kernel[i] /= kernelSum;
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Executar per X
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t0, 0);
    convolutionXKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionXKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "X filter: " << t01 << " ms" << std::endl;

    // Copiar el resultat a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Y
    hipEventRecord(t0, 0);
    convolutionYKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionYKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "Y filter: " << t01 << " ms" << std::endl;

    // Copiar el resultat a l'array
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Z
    hipEventRecord(t0, 0);
    convolutionZKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionZKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "Z filter: " << t01 << " ms" << std::endl;

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Resta
    hipEventRecord(t0, 0);
    substractionKernel<<<blockGrid, threadBlock>>>(dfResult, volumeDataDims);
    CUT_CHECK_ERROR( "substractionKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(t1, 0);
    hipEventSynchronize(t1);
    hipEventElapsedTime(&t01, t0, t1);
    std::cout << "substraction: " << t01 << " ms" << std::endl;

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "gaussian difference: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(t0);
    hipEventDestroy(t1);

    return result;
}


QVector<float> cfBoxMeanDifference(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Executar per X
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    convolutionXKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionXKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Y
    convolutionYKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionYKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat a l'array
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Executar per Z
    convolutionZKernel<<<blockGrid, threadBlock>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    CUT_CHECK_ERROR( "convolutionZKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Resta
    substractionKernel<<<blockGrid, threadBlock>>>(dfResult, volumeDataDims);
    CUT_CHECK_ERROR( "substractionKernel() execution failed\n" );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "box mean difference: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void squareKernel(float *result, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] = value * value;
}


__global__ void finalChebychevKernel(float *result, float *result2, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);

    uint i = x + y * dims.width + z * dims.width * dims.height;

    float mean = result[i];         // E[Z]

    if (value > mean)
    {
        float squaresMean = result2[i]; // E[Z²]

        float variance = squaresMean - mean * mean;
        float a = value - mean;  // z - E[Z]
        result[i] = variance / (variance + a * a);  // Chebychev inequality
    }
    else result[i] = 1.0f;
}


QVector<float> cfProbabilisticAmbientOcclusionGaussianChebychev(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    float kernelSum = 0.0f;
    float sigma = radius / 3.0f;
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        float f = static_cast<float>(i - radius) / sigma;
        kernel[i] = expf(-f * f / 2.0f);
        kernelSum += kernel.at(i);
    }
    for (int i = 0; i < KERNEL_WIDTH; i++) kernel[i] /= kernelSum;
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // aquesta la fem síncrona
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalChebychevKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "pao gaussian chebychev: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


QVector<float> cfProbabilisticAmbientOcclusionBoxMeanChebychev(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // aquesta la fem síncrona
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalChebychevKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "pao box mean chebychev: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void finalGaussianKernel(float *result, float *result2, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);

    uint i = x + y * dims.width + z * dims.width * dims.height;

    float mean = result[i];         // E[Z]
    float squaresMean = result2[i]; // E[Z²]
    float variance = squaresMean - mean * mean;
    // Fórmules tretes de:
    // http://en.wikipedia.org/wiki/Gaussian_distribution#Cumulative_distribution_function
    // http://en.wikipedia.org/wiki/Q-function
    // P(Z >= z) = 1/2 * erfc((z - mean) / sqrt(2 * variance))
    result[i] = 0.5f * erfcf((value - mean) / sqrtf(2.0f * variance));
}


QVector<float> cfProbabilisticAmbientOcclusionGaussian(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el volum original un altre cop a l'array
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyHostToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // aquesta la fem síncrona
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalGaussianKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "pao gaussian: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void cubeFrequencyKernel(float *result, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    int greater = 0;    // compta quants n'hi ha de més grans o iguals que value al seu entorn

    for (int dx = -radius; dx <= radius; dx++)
    {
        float fx2 = fx + dx;

        for (int dy = -radius; dy <= radius; dy++)
        {
            float fy2 = fy + dy;

            for (int dz = -radius; dz <= radius; dz++)
            {
                float fz2 = fz + dz;
                float value2 = tex3D(gVolumeTexture, fx2, fy2, fz2);
                if (value2 >= value) greater++;
            }
        }
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;
    int length = 2 * radius + 1;
    float count = length * length * length;
    result[i] = greater / count;
}


QVector<float> cfProbabilisticAmbientOcclusionCube(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Llancem el kernel
    cubeFrequencyKernel<<<blockGrid, threadBlock>>>(dfResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfResult) );;
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "pao cube: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void sphereFrequencyKernel(float *result, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolumeTexture, fx, fy, fz);
    int greater = 0;    // compta quants n'hi ha de més grans o iguals que value al seu entorn
    int count = 0;

    for (int dx = -radius; dx <= radius; dx++)
    {
        float fx2 = fx + dx;

        for (int dy = -radius; dy <= radius; dy++)
        {
            if (sqrtf(dx * dx + dy * dy) > radius) continue;

            float fy2 = fy + dy;

            for (int dz = -radius; dz <= radius; dz++)
            {
                if (sqrtf(dx * dx + dy * dy + dz * dz) > radius) continue;

                float fz2 = fz + dz;
                float value2 = tex3D(gVolumeTexture, fx2, fy2, fz2);
                if (value2 >= value) greater++;
                count++;
            }
        }
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] = ((float) greater) / count;
}


QVector<float> cfProbabilisticAmbientOcclusionSphere(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Llancem el kernel
    sphereFrequencyKernel<<<blockGrid, threadBlock>>>(dfResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfResult) );;
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "pao sphere: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void finalTangentSphereVarianceKernel(float *result, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolume3Texture, fx, fy, fz);
    float3 normal = normalize(make_float3(tex3D(gVolume3Texture, fx + 1.0f, fy, fz) - tex3D(gVolume3Texture, fx - 1.0f, fy, fz),
                                          tex3D(gVolume3Texture, fx, fy + 1.0f, fz) - tex3D(gVolume3Texture, fx, fy - 1.0f, fz),
                                          tex3D(gVolume3Texture, fx, fy, fz + 1.0f) - tex3D(gVolume3Texture, fx, fy, fz - 1.0f)));

    float3 c = normal * radius / 2.0f;
    float cx = fx + c.x, cy = fy + c.y, cz = fz + c.z;

    float mean = tex3D(gVolumeTexture, cx, cy, cz); // E[Z]

    uint i = x + y * dims.width + z * dims.width * dims.height;

    if (value > mean)
    {
        float squaresMean = tex3D(gVolume2Texture, cx, cy, cz); // E[Z²]
        float variance = squaresMean - mean * mean;
        float a = value - mean;  // z - E[Z]
        result[i] = variance / (variance + a * a);
    }
    else result[i] = 0.0f;
}


// implementació barroera
QVector<float> cfProbabilisticAmbientOcclusionTangentSphereVariance(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolume3Array;
    hipChannelFormatDesc channelDescVolume3Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume3Array, &channelDescVolume3Array, volumeDataDims) );
    hipMemcpy3DParms copyParams3 = {0};
    copyParams3.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams3.dstArray = dVolume3Array;
    copyParams3.extent = volumeDataDims;
    copyParams3.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams3) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolume3Texture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume3Texture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume3Texture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume3Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume3Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume3Texture, dVolume3Array, channelDescVolumeArray) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalTangentSphereVarianceKernel<<<blockGrid, threadBlock>>>(dfResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume3Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume3Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "paots variance: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


// separem els mínims i màxims en un volum i les mitjanes en un altre perquè CUDA no accepta volums de float3
texture<float2, 3> gMinMaxVolumeTexture;    // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)
texture<float, 3> gMeanVolumeTexture;       // el 3r paràmetre pot ser hipReadModeElementType (valor directe) (predeterminat) o hipReadModeNormalizedFloat (valor escalat entre 0 i 1)


// separem els mínims i màxims en un volum i les mitjanes en un altre perquè CUDA no accepta volums de float3
__global__ void minMaxMeanFilteringXKernel(float2 *minMax, float *mean, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float min = HIP_INF_F, max = 0.0f, sum = 0.0f;

    for (int k = -radius; k <= radius; k++)
    {
        float value = tex3D(gVolumeTexture, fx + k, fy, fz);
        min = fminf(min, value);
        max = fmaxf(max, value);
        sum += value;
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    minMax[i].x = min;
    minMax[i].y = max;
    mean[i] = sum / (2 * radius + 1);
}


// separem els mínims i màxims en un volum i les mitjanes en un altre perquè CUDA no accepta volums de float3
__global__ void minMaxMeanFilteringYKernel(float2 *minMax, float *mean, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float min = HIP_INF_F, max = 0.0f, sum = 0.0f;

    for (int k = -radius; k <= radius; k++)
    {
        float2 value = tex3D(gMinMaxVolumeTexture, fx, fy + k, fz);
        min = fminf(min, value.x);
        max = fmaxf(max, value.y);
        sum += tex3D(gMeanVolumeTexture, fx, fy + k, fz);
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    minMax[i].x = min;
    minMax[i].y = max;
    mean[i] = sum / (2 * radius + 1);
}


// separem els mínims i màxims en un volum i les mitjanes en un altre perquè CUDA no accepta volums de float3
__global__ void minMaxMeanFilteringZKernel(float2 *minMax, float *mean, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float min = HIP_INF_F, max = 0.0f, sum = 0.0f;

    for (int k = -radius; k <= radius; k++)
    {
        float2 value = tex3D(gMinMaxVolumeTexture, fx, fy, fz + k);
        min = fminf(min, value.x);
        max = fmaxf(max, value.y);
        sum += tex3D(gMeanVolumeTexture, fx, fy, fz + k);
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;

    minMax[i].x = min;
    minMax[i].y = max;
    mean[i] = sum / (2 * radius + 1);
}


// separem els mínims i màxims en un volum i les mitjanes en un altre perquè CUDA no accepta volums de float3
__global__ void finalTangentSphereCdfKernel(float *result, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;

    float vstar = tex3D(gVolumeTexture, fx, fy, fz);
    float3 normal = normalize(make_float3(tex3D(gVolumeTexture, fx + 1.0f, fy, fz) - tex3D(gVolumeTexture, fx - 1.0f, fy, fz),
                                          tex3D(gVolumeTexture, fx, fy + 1.0f, fz) - tex3D(gVolumeTexture, fx, fy - 1.0f, fz),
                                          tex3D(gVolumeTexture, fx, fy, fz + 1.0f) - tex3D(gVolumeTexture, fx, fy, fz - 1.0f)));
    float3 c = normal * radius / 2.0f;
    float cx = fx + c.x, cy = fy + c.y, cz = fz + c.z;
    float2 minMax = tex3D(gMinMaxVolumeTexture, cx, cy, cz);
    float vmin = minMax.x;
    float vmax = minMax.y;
    float mean = tex3D(gMeanVolumeTexture, cx, cy, cz);
    float Dv = vmax - vmin;

    float a2 = 3 * (2 * mean - vmax - vmin) / Dv;
    if (a2 < -1) a2 = -1;
    if (a2 > 1) a2 = 1;

    float a1 = 1 - a2;

    float ao = 1;   // obscurance
    if (vstar < vmin) ao = 0;
    else if (vstar < vmax )
    {
        float t = (vstar - vmin) / Dv;
        ao = a2 * t * t + a1 * t;
    }

    uint i = x + y * dims.width + z * dims.width * dims.height;
    result[i] = 1 - ao;
}


// separem els mínims i màxims en un volum i les mitjanes en un altre perquè CUDA no accepta volums de float3
QVector<float> cfProbabilisticAmbientOcclusionTangentSphereCdf( vtkImageData *image, int radius )
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Crear un array pel volum de mínims i màxims, amb la seva textura corresponent
    hipArray *dMinMaxVolumeArray;
    hipChannelFormatDesc channelDescMinMaxVolumeArray = hipCreateChannelDesc<float2>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dMinMaxVolumeArray, &channelDescMinMaxVolumeArray, volumeDataDims) );
    //gMinMaxVolumeTexture.normalized = false;                    // false (predeterminat) -> [0,N) | true -> [0,1)
    //gMinMaxVolumeTexture.filterMode = hipFilterModePoint;      // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gMinMaxVolumeTexture.addressMode[0] = hipAddressModeClamp; // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gMinMaxVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gMinMaxVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gMinMaxVolumeTexture, dMinMaxVolumeArray, channelDescMinMaxVolumeArray) );

    // Reservar espai pels resultats intermitjos de mínims i màxims del filtratge
    float2 *dfMinMaxResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfMinMaxResult), VOLUME_DATA_SIZE * sizeof(float2)) );

    // Crear un array pel volum de mitjanes, amb la seva textura corresponent
    hipArray *dMeanVolumeArray;
    hipChannelFormatDesc channelDescMeanVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dMeanVolumeArray, &channelDescMeanVolumeArray, volumeDataDims) );
    //gMeanVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gMeanVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gMeanVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gMeanVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gMeanVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gMeanVolumeTexture, dMeanVolumeArray, channelDescMeanVolumeArray) );

    // Reservar espai pels resultats intermitjos de mitjanes del filtratge
    float *dfMeanResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfMeanResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Executar per X
    minMaxMeanFilteringXKernel<<<blockGrid, threadBlock>>>(dfMinMaxResult, dfMeanResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar els resultats als arrays
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfMinMaxResult), dimensions[0] * sizeof(float2), dimensions[0], dimensions[1]); // data, pitch, width, height
    copyParams.dstArray = dMinMaxVolumeArray;
    copyParams.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfMeanResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dMeanVolumeArray;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );

    // Executar per Y
    minMaxMeanFilteringYKernel<<<blockGrid, threadBlock>>>(dfMinMaxResult, dfMeanResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar els resultats als arrays
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfMinMaxResult), dimensions[0] * sizeof(float2), dimensions[0], dimensions[1]); // data, pitch, width, height
    copyParams.dstArray = dMinMaxVolumeArray;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfMeanResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dMeanVolumeArray;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );

    // Executar per Z
    minMaxMeanFilteringZKernel<<<blockGrid, threadBlock>>>(dfMinMaxResult, dfMeanResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar els resultats als arrays
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfMinMaxResult), dimensions[0] * sizeof(float2), dimensions[0], dimensions[1]); // data, pitch, width, height
    copyParams.dstArray = dMinMaxVolumeArray;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfMeanResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dMeanVolumeArray;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );

    // Neteja parcial
    CUDA_SAFE_CALL( hipFree(dfMinMaxResult) );
    CUDA_SAFE_CALL( hipFree(dfMeanResult) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Ara ja podem fer la passada final
    finalTangentSphereCdfKernel<<<blockGrid, threadBlock>>>(dfResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gMinMaxVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gMeanVolumeTexture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dMinMaxVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dMeanVolumeArray) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "paots cdf: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


__global__ void finalTangentSphereGaussianKernel(float *result, int radius, hipExtent dims)
{
    uint blocksX = iDivUp(dims.width, blockDim.x);
    uint blockX = blockIdx.x % blocksX;
    uint blockY = blockIdx.x / blocksX;
    uint blockZ = blockIdx.y;

    uint x = blockX * blockDim.x + threadIdx.x;
    if (x >= dims.width) return;
    uint y = blockY * blockDim.y + threadIdx.y;
    if (y >= dims.height) return;
    uint z = blockZ * blockDim.z + threadIdx.z;
    if (z >= dims.depth) return;

    float fx = x + 0.5f, fy = y + 0.5f, fz = z + 0.5f;
    float value = tex3D(gVolume3Texture, fx, fy, fz);
    float3 normal = normalize(make_float3(tex3D(gVolume3Texture, fx + 1.0f, fy, fz) - tex3D(gVolume3Texture, fx - 1.0f, fy, fz),
                                          tex3D(gVolume3Texture, fx, fy + 1.0f, fz) - tex3D(gVolume3Texture, fx, fy - 1.0f, fz),
                                          tex3D(gVolume3Texture, fx, fy, fz + 1.0f) - tex3D(gVolume3Texture, fx, fy, fz - 1.0f)));

    float3 c = normal * radius / 2.0f;
    float cx = fx + c.x, cy = fy + c.y, cz = fz + c.z;

    float mean = tex3D(gVolumeTexture, cx, cy, cz); // E[Z]
    float squaresMean = tex3D(gVolume2Texture, cx, cy, cz); // E[Z²]
    float variance = squaresMean - mean * mean;
    if (variance > -0.001f && variance < 0.0f) variance = 0.0f; // per evitar nans

    uint i = x + y * dims.width + z * dims.width * dims.height;

    // Fórmules tretes de:
    // http://en.wikipedia.org/wiki/Gaussian_distribution#Cumulative_distribution_function
    // http://en.wikipedia.org/wiki/Q-function
    // P(Z >= z) = 1/2 * erfc((z - mean) / sqrt(2 * variance))
    result[i] = 0.5f * erfcf((value - mean) / sqrtf(2.0f * variance));
}


QVector<float> cfProbabilisticAmbientOcclusionTangentSphereGaussian(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);
    
    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolume3Array;
    hipChannelFormatDesc channelDescVolume3Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume3Array, &channelDescVolume3Array, volumeDataDims) );
    hipMemcpy3DParms copyParams3 = {0};
    copyParams3.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams3.dstArray = dVolume3Array;
    copyParams3.extent = volumeDataDims;
    copyParams3.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams3) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolume3Texture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume3Texture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume3Texture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume3Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume3Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume3Texture, dVolume3Array, channelDescVolumeArray) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Ara ja podem fer la passada final
    finalTangentSphereGaussianKernel<<<blockGrid, threadBlock>>>(dfResult, radius, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume3Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume3Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "paots gaussian: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}


QVector<float> cfVolumeVariance(vtkImageData *image, int radius)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *data = reinterpret_cast<float*>(image->GetScalarPointer());
    const uint VOLUME_DATA_SIZE = image->GetNumberOfPoints();
    int *dimensions = image->GetDimensions();
    hipExtent volumeDataDims = make_hipExtent(dimensions[0], dimensions[1], dimensions[2]);

    // Copiar el volum a un array i associar-hi una textura
    hipArray *dVolumeArray;
    hipChannelFormatDesc channelDescVolumeArray = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolumeArray, &channelDescVolumeArray, volumeDataDims) );
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(data), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.dstArray = dVolumeArray;
    copyParams.extent = volumeDataDims;
    copyParams.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );    // còpia síncrona perquè si un dels dos és el host ha de ser memòria reservada amb hipHostMalloc
    //gVolumeTexture.normalized = false;                      // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolumeTexture.filterMode = hipFilterModePoint;        // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolumeTexture.addressMode[0] = hipAddressModeClamp;   // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolumeTexture.addressMode[1] = hipAddressModeClamp;
    //gVolumeTexture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolumeTexture, dVolumeArray, channelDescVolumeArray) );

    // Reservar espai pel resultat
    float *dfResult;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult), VOLUME_DATA_SIZE * sizeof(float)) );

    // Preparar l'execució
    //Block width should be a multiple of maximum coalesced write size
    //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
    dim3 threadBlock(16, 8, 4);
    uint blocksX = iDivUp(volumeDataDims.width, threadBlock.x);
    uint blocksY = iDivUp(volumeDataDims.height, threadBlock.y);
    uint blocksZ = iDivUp(volumeDataDims.depth, threadBlock.z);
    dim3 blockGrid(blocksX * blocksY, blocksZ);

    // Calcular volum al quadrat
    squareKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult, volumeDataDims); // generem el volum al quadrat a l'stream 2

    // Mentrestant, al host...

    // Crear un segon array pel volum al quadrat, amb la seva textura corresponent
    hipArray *dVolume2Array;
    hipChannelFormatDesc channelDescVolume2Array = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL( hipMalloc3DArray(&dVolume2Array, &channelDescVolume2Array, volumeDataDims) );
    //gVolume2Texture.normalized = false;                     // false (predeterminat) -> [0,N) | true -> [0,1)
    //gVolume2Texture.filterMode = hipFilterModePoint;       // hipFilterModePoint (predeterminat) o hipFilterModeLinear
    //gVolume2Texture.addressMode[0] = hipAddressModeClamp;  // hipAddressModeClamp (retallar) (predeterminat) o hipAddressModeWrap (fer la volta)
    //gVolume2Texture.addressMode[1] = hipAddressModeClamp;
    //gVolume2Texture.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL( hipBindTextureToArray(gVolume2Texture, dVolume2Array, channelDescVolume2Array) );

    // En aquest temps suposem que ja ha acabat el kernel d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // Copiem el resultat a l'array del volum al quadrat
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);   // data, pitch, width, height
    copyParams2.dstArray = dVolume2Array;
    copyParams2.extent = volumeDataDims;
    copyParams2.kind = hipMemcpyDeviceToDevice;
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) ); // còpia a l'stream 2

    // Reservar espai per l'altre resultat
    float *dfResult2;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfResult2), VOLUME_DATA_SIZE * sizeof(float)) );

    // Calcular kernel
    const int KERNEL_WIDTH = 2 * radius + 1;
    QVector<float> kernel(KERNEL_WIDTH);
    kernel.fill(1.0f / KERNEL_WIDTH);
    std::cout << "kernel:";
    for (int i = 0; i < KERNEL_WIDTH; i++) std::cout << " " << kernel[i];
    std::cout << std::endl;
    float *dfKernel;
    CUDA_SAFE_CALL( hipMalloc(reinterpret_cast<void**>(&dfKernel), KERNEL_WIDTH * sizeof(float)) );
    // Aquest és molt petit, o sigui que el podem fer síncron
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(dfKernel), reinterpret_cast<void*>(kernel.data()), KERNEL_WIDTH * sizeof(float), hipMemcpyHostToDevice) );

    // Aquí ja hauria d'haver acabat la còpia d'abans (stream 2)
    hipStreamSynchronize(stream2);

    // A partir d'aquí comença la diversió:
    // Executarem els dos filtratges en paral·lel, en streams diferents, perquè mentre un filtra l'altre copiï memòria i viceversa.
    // Així podem aconseguir una execució més ràpida (en teoria).
    // Som-hi doncs...

    // Executar per X1
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per X2
    convolutionXKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    copyParams.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);    // data, pitch, width, height
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    copyParams2.srcPtr = make_hipPitchedPtr(reinterpret_cast<void*>(dfResult2), dimensions[0] * sizeof(float), dimensions[0], dimensions[1]);  // data, pitch, width, height
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Y1
    hipStreamSynchronize(stream1);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Y2
    hipStreamSynchronize(stream2);
    convolutionYKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Copiar el resultat a l'array (1)
    hipStreamSynchronize(stream1);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams, stream1) );
    // Copiar el resultat a l'array (2)
    hipStreamSynchronize(stream2);
    CUDA_SAFE_CALL_NO_SYNC( hipMemcpy3DAsync(&copyParams2, stream2) );

    // Executar per Z1
    hipStreamSynchronize(stream1);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream1>>>(dfResult, dfKernel, radius, volumeDataDims, false);
    // Executar per Z2
    hipStreamSynchronize(stream2);
    convolutionZKernel<<<blockGrid, threadBlock, 0, stream2>>>(dfResult2, dfKernel, radius, volumeDataDims, true);

    // Ara ja podem fer la passada final
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    finalChebychevKernel<<<blockGrid, threadBlock>>>(dfResult, dfResult2, volumeDataDims);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // Copiar el resultat final al host
    QVector<float> result(VOLUME_DATA_SIZE);
    CUDA_SAFE_CALL( hipMemcpy(reinterpret_cast<void*>(result.data()), reinterpret_cast<void*>(dfResult), VOLUME_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    // Neteja
    CUDA_SAFE_CALL( hipFree(dfKernel) );
    CUDA_SAFE_CALL( hipFree(dfResult) );
    CUDA_SAFE_CALL( hipFree(dfResult2) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolumeTexture) );
    CUDA_SAFE_CALL( hipUnbindTexture(gVolume2Texture) );
    CUDA_SAFE_CALL( hipFreeArray(dVolumeArray) );
    CUDA_SAFE_CALL( hipFreeArray(dVolume2Array) );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "volume variance: " << elapsedTime << " ms" << std::endl;

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return result;
}
